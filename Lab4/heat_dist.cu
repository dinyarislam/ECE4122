/*
Author: Dinyar Islam
Class: ECE4122
Last Date Modified: 10 November 2022

Description: Lab 4

What is the purpose of this file?
CUDA program to determine the steady state heat distribution in a thin metal plate using synchronous iteration on a GPU.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <string>
#include <iostream>
#include <fstream>

using namespace std;

/*
The HANDLE_ERROR method
@param result
*/
inline hipError_t HANDLE_ERROR(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        assert(result == cudaSuccess);
    }
#endif
    return result;
}

/*
The calculateTemp method is the kernal for calculating the temperature on each thread
@param h the memory address to the h array
@param g the memory address to the g array
@param limit the maximum number of iterations
@param n number of interior points 
*/
__global__
void calculateTemp(double* h, double* g, int n, int limit)
{
    // Calculate the column index of the Pd element, denote by x
    int y = threadIdx.x + blockIdx.x * blockDim.x;
    // Calculate the row index of the Pd element, denote by y
    int x = threadIdx.y + blockIdx.y * blockDim.y;

    // exit kernal if exterior point
    if (x <= 0 || x >= n+1 || y <= 0 || y >= n+1)
    {
        return;
    }
    else
    {
        g[y * (n+2) + x] = 0.25 * (h[(y-1) * (n+2) + x] + h[(y+1) * (n+2) + x] + h[y * (n+2) + x-1] + h[y * (n+2) + x+1]);
    }
}


/*
The main method that uses CUDA to solve the steady state temperature by calling the kernal above
@param argc the number of command-line arguments
@param *argv[] pointer to array of arguments
*/
int main(int argc, char* argv[])
{
    // validation check for argument count
    if (argc != 5)
    {
        std::cout << "Invalid Input!" << std::endl;
        return 0;
    }

    // extract cmd line arguments
    std::string argStr_n;
    std::string argStr_limit;
    std::string arg_n = "-n";
    std::string arg_I = "-I";

    if (arg_n.compare(argv[1]) == 0 && arg_I.compare(argv[3]) == 0)
    {
        argStr_n = argv[2];
        argStr_limit = argv[4];
    }
    else if (arg_I.compare(argv[1]) == 0 && arg_n.compare(argv[3]) == 0)
    {
        argStr_n = argv[4];
        argStr_limit = argv[2];
    }
    else
    {
        std::cout << "Invalid Input!" << std::endl;
        return 0;
    }

    // std::cout<<argStr_n<<"\n";
    // std::cout<<argStr_limit<<"\n";

    // validation check for argument values
    for (int j = 0; j < argStr_n.length(); j++)
    {
        if (!isdigit(argStr_n[j])) //check if input has ASCII characters other than digits
        {
            std::cout << "Invalid Input!" << std::endl;
            return 0;
        }
    }
    for (int j = 0; j < argStr_limit.length(); j++)
    {
        if (!isdigit(argStr_limit[j])) //check if input has ASCII characters other than digits
        {
            std::cout << "Invalid Input!" << std::endl;
            return 0;
        }
    }
    int n = std::stoi(argStr_n);
    int limit = std::stoi(argStr_limit);
    if (n <= 0 | limit <= 0)
    {
        std::cout << "Incorrect value for n or I" << "\n";
        return 0;
    }


    // Use a 1D array to represent a two dimensional array
    int size = (n + 2) * (n + 2) * sizeof(double);
    double* h;
    double* g;

    // capture start time
    hipEvent_t     start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    // allocate unified memory on the CPU and GPU (more streamlined than as device to host copying not required)
    HANDLE_ERROR(hipMallocManaged((void**) &h, size));
    HANDLE_ERROR(hipMallocManaged((void**) &g, size));

    // initialize the arrays
    for (int y = 0; y < n+2; y++)
    {
        for (int x = 0; x < n+2; x++)
        {
            if (y == 0 | x == 0 | y == n + 1 | x == n + 1)
            {
                if (y == 0 & (x >= std::round(0.3 * (n + 2))) & (x < std::round(0.7 * (n + 2))))
                {
                    h[y * (n+2) + x] = 100;
                    g[y * (n+2) + x] = 100;
                }
                else
                {
                    h[y * (n+2) + x] = 20;
                    g[y * (n+2) + x] = 20;
                }
            }
            else
            {
                h[y * (n+2) + x] = 0;
                g[y * (n+2) + x] = 0;
            }
        }
    }

    // kernel invocation code
    dim3 dimBlock(32, 32);
    dim3 dimGrid((n + 34) / 32, (n + 34) / 32);

    // std::cout<<n<<endl;
    // std::cout<<limit<<endl;

    // alternate between h and g to allow array copying
    for (int i = 0; i < limit; i++)
    {
        if (i % 2)
        {
            // for (int i = 0; i < (n + 2); i++)
            // {
            //     for (int j = 0; j < (n + 2); j++)
            //     {
            //         std::cout<<h[i*(n+2)+j]<<",";
            //     }
            //     std::cout<<"\n";
            // }
            //std::cout<<"hello"<<"\n";
            calculateTemp<<<dimGrid, dimBlock>>>(g, h, n, limit);
        }
        else
        {
            calculateTemp<<<dimGrid, dimBlock>>>(h, g, n, limit);
        }
        // Wait for GPU to finish before accessing on host
        hipDeviceSynchronize();
    }

    // get stop time, and display the timing results
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float   elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("Thin plate calculation took  %.3f milliseconds\n", elapsedTime);

    // destroy events to free memory
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));


    // csv file output
    std::fstream fout;
    fout.open("finalTemperatures.csv", std::ios::out);
    for (int i = 0; i < n + 2; i++)
    {
        for (int j = 0; j < n + 2; j++)
        {
            fout << h[(i * (n + 2)) + j] << ", ";
        }
        fout << "\n";
        //std::cout<<"hello"<<"\n";
    }

    // free the memory allocated on the GPU
    HANDLE_ERROR(hipFree(h));
    HANDLE_ERROR(hipFree(g));

    return 0;
}

/*
PACE-ICE CMD:
module load gcc/9.2.0 cuda/11.1
nvcc *.cu -o main
*/